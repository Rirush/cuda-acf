
#include <hip/hip_runtime.h>
#include <cstdio>
#include <cinttypes>
#include <chrono>
#include <bitset>

#define BLOCK_SIZE 64

// Задача 1. Определить бит для каждой нити с учетом того, что каждой нити прихоидтся N (около 3-5, больше 5 может отсутствовать какой-либо выигрыш) чисел
// Задача 2. Расчитать АКФ для каждого числа
// Задача 3. Запись результата в глобальную память одна на блок, наилучший результат
//
// Можно использовать атомики чтобы писать в глобальную память только одно значение
// Начальный пик всегда ярко выражен, поэтому его учитывать не надо
// Париться о длинной арифметике не надо - больше 32-33 бит посчитать не получится в адекватное время

__global__ void akf(uint64_t start_offset, uint64_t end, uint64_t *min_amplitude, uint64_t *signal, uint64_t n) {
    __shared__ int8_t akf[BLOCK_SIZE];
    __shared__ uint8_t bits[BLOCK_SIZE];
    __shared__ uint64_t amplitude;
    size_t idx = threadIdx.x; // offset
    uint64_t val = blockIdx.x + start_offset; // signal
    while (val <= end) {
        amplitude = 0;
        bits[n - idx - 1] = (val >> idx) & 1;
        __syncthreads();

        int8_t out = 0;
        for (size_t i = 0; i < n - idx; i++) {
            out -= (bits[i + idx] ^ bits[i]) * 2 - 1;
        }

        akf[idx] = (out >= 0) ? out : 0 - out;
        if (idx != 0) {
            atomicMax(reinterpret_cast<unsigned long long int*>(&amplitude), (unsigned long long)akf[idx]);
        }
        __syncthreads();

        if (idx == 0) {
            uint64_t old = atomicMin(reinterpret_cast<unsigned long long int*>(min_amplitude), (unsigned long long)amplitude);
            if (old >= amplitude) {
                *signal = val;
            }
        }

        val += gridDim.x;
        __syncthreads();
    };
}

void cpu_akf(uint64_t value, uint8_t b, int8_t *akf) {
    auto bits = new int8_t[b]();
    for(size_t i = 0; i < b; i++) {
        int8_t bit = (value & 1) ? 1 : -1;
        bits[b - i - 1] = bit;
        value >>= 1;
    }

    for(size_t offset = 0; offset < b; offset++) {
        for(size_t pos = 0; pos + offset < b; pos++) {
            akf[offset] += bits[pos] * bits[pos + offset];
        }
    }
    delete[] bits;
}

int main(int argc, char **argv) {
    if(argc != 2) {
        printf("Usage: analysis [size]\n");
        return EXIT_FAILURE;
    }
    int n = atoi(argv[1]);
    printf("Using %d bits\n", n);
    if(n >= 64) {
        printf("Cannot process more than 63 bits\n");
        return EXIT_FAILURE;
    }
    if(n < 5) {
        printf("Cannot process less than 5 bits\n");
        return EXIT_FAILURE;
    }
    auto start_time = std::chrono::high_resolution_clock::now();
    uint64_t *dev_amplitude;
    uint64_t *dev_signal;
    hipMalloc((void**)&dev_amplitude, sizeof(uint64_t));
    hipMalloc((void**)&dev_signal, sizeof(uint64_t));
    hipMemset(dev_amplitude, 0xFF, sizeof(uint64_t));
    uint64_t amplitude;
    uint64_t signal;

    uint64_t blocks = (1ULL << n) - 1ULL;
    uint64_t start = 0b1ULL << (n - 1);
    akf<<<3072, n>>>(start, blocks, dev_amplitude, dev_signal, n);
    hipError_t hipError_t = hipGetLastError();
    if(hipError_t != hipSuccess) {
        printf("Could not execute CUDA kernel: %s\n", hipGetErrorString(hipError_t));
        return EXIT_FAILURE;
    }
    hipMemcpy(&amplitude, dev_amplitude, sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(&signal, dev_signal, sizeof(uint64_t), hipMemcpyDeviceToHost);
    std::bitset<BLOCK_SIZE> s(signal);
    printf("Best signal is %s (%llu) with amplitude of %llu\n", s.to_string().c_str(), signal, amplitude);

    auto *akf = new int8_t[n]();
    cpu_akf(signal, n, akf);
    printf("AKF is ");
    for(int8_t i = 0; i < n; i++) {
        printf("%d ", akf[i]);
    }
    printf("\n");
    auto end_time = std::chrono::high_resolution_clock::now();
    printf("Calculation took %f seconds", (double)(std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count()) / 1e+6);
    return 0;
}
